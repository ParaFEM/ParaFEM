
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hipblas.h>

/* Kernel definitions */

/* 
  This kernel assumes matrix is stored column wise

  This attempts 2D thread blocks

  Things to add: shared memory for temp array

*/
/* __global__ void MultiMatVecMultiply3(int n_mat, */
/* 				     int n_row, */
/* 				     int n_col, */
/* 				     double* lhs_vector, */
/* 				     double* matrix, */
/* 				     double* rhs_vector) */
/* { */
/*   int global_row_id; */
/*   int matrix_id; */
/*   int row_id; */
/*   int j;  */
/*   double tmp; */

/*   /\* Get the global index that corresponds to a row of some matrix *\/ */
/*   global_row_id =  threadIdx.x + blockIdx.x * blockDim.x;   */
  
/*   if (global_row_id < n_mat*n_row) */
/*     { */
/*       /\* get matrix id *\/ */
/*       matrix_id = global_row_id/n_row; */
      
/*       /\* Get local row id *\/ */
/*       row_id = global_thread_id%n_row;  */
      
/*       /\* Change tmp to be a shared array of size *\/ */
/*       tmp = 0.0; */
      
/*       /\* Change this loop to use y dim thread *\/ */
/*       for (j=threadIdx.y; j<n_col; j+= blockDim.y) */
/* 	{ */
/* 	  tmp += */
/* 	    lhs_vector[j+matrix_id*n_col] *  */
/* 	    matrix[row_id+j*n_row+matrix_id*n_col*n_row]; */
/* 	} */

/*       /\* Reduce sums *\/ */

/*       /\* Copy reduced value to result vector *\/ */
/*       rhs_vector[global_thread_id] = tmp; */
/*     } */
/* } */

/* This kernel used shared memory but assumes:

   - the matrix is stored column wise 
   - the 1D thread block is of size 60
   - 60x60 matrix

*/
__global__ void MultiMatVecMultiply2(int n_mat,
				     int n_row,
				     int n_col,
				     double* lhs_vector,
				     double* matrix,
				     double* rhs_vector)
{
  int local_thread_id;
  int block_id;
  int matrix_id;
  int global_thread_id;
  int row_id;
  int j; 
  double tmp;
  
  /* Get 1D thread index */
  local_thread_id = threadIdx.x;
  
  /* Get 1D block index */
  block_id = blockIdx.x;

  /* Global thread index */
  global_thread_id = local_thread_id + block_id * blockDim.x; 

  /* get matrix id */
  matrix_id = global_thread_id/n_row;

  /* Shared memory to store copy of lhs vector */
  __shared__ double lhs_vector_shared[60];
  
  /* Get row id */
  row_id = global_thread_id%n_row; 
  
  /* Load values into shared vector */
  if (local_thread_id < 60)
    lhs_vector_shared[local_thread_id] = 
      lhs_vector[matrix_id*n_col+local_thread_id];

  /* Sunc threads */
  __syncthreads();

  if (global_thread_id < n_mat*n_row)
    {
      tmp = 0.0;
      for (j=0; j<n_col; j++)
	{
	  tmp +=
	    lhs_vector_shared[j] * 
	    matrix[j*n_row+row_id+matrix_id*n_col*n_row];
	}
      rhs_vector[global_thread_id] = tmp;
    }
}

/* 
  This kernel assumes matrix is stored column wise

  This is a naive implementation, no attempt at optimisation
*/
__global__ void MultiMatVecMultiply1(int n_mat,
				     int n_row,
				     int n_col,
				     double* lhs_vector,
				     double* matrix,
				     double* rhs_vector)
{
  int local_thread_id;
  int block_id;
  int matrix_id;
  int global_thread_id;
  int row_id;
  int j; 
  double tmp;
  
  /* Get 1D thread index */
  local_thread_id = threadIdx.x;
  
  /* Get 1D block index */
  block_id = blockIdx.x;

  global_thread_id = local_thread_id + block_id * blockDim.x;  
  
  if (global_thread_id < n_mat*n_row)
    {
      
      /* get matrix id */
      matrix_id = global_thread_id/n_row;
      
      /* Get row id */
      row_id = global_thread_id%n_row; 
      
      tmp = 0.0;
      for (j=0; j<n_col; j++)
	{
	  tmp +=
	    lhs_vector[j+matrix_id*n_col] * 
	    matrix[row_id+j*n_row+matrix_id*n_col*n_row];
	}
      rhs_vector[global_thread_id] = tmp;
    }
}


/* Helper functions */

/* Function to allocate memory on device */
extern "C" int allocate_memory_on_gpu(const int *n_elements, 
				      const int *element_size,
				      void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipMalloc(device_pointer, *element_size * *n_elements);
  
  if (cuda_status != hipSuccess)
    {
      printf("Device memory failed to allocate!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}

/* Function to free memory on device */
extern "C" int free_memory_on_gpu(void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipFree(*device_pointer);
  
  if (cuda_status != hipSuccess)
    {
      printf("Device memory failed to deallocate!\n");
      return EXIT_FAILURE;
    }
  
  return EXIT_SUCCESS;
}

/* Function to copy data to the gpu */
extern "C" int copy_data_to_gpu(const int *n_elements,
				const int *element_size,
				const void *host_data,
				void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipMemcpy(*device_pointer,
			   host_data,
			   *n_elements * *element_size,
			   hipMemcpyHostToDevice);
  
  if (cuda_status != hipSuccess)
    {
      printf("Failed to copy data to device!\n");
      return EXIT_FAILURE;
    }
  
  return EXIT_SUCCESS;
 }

/* Function to copy data from the gpu */
extern "C" int copy_data_from_gpu(const int *n_elements, 
				  const int *element_size,
				  void *host_data,
				  void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipMemcpy(host_data,
			   *device_pointer,
			   *n_elements * *element_size,
			   hipMemcpyDeviceToHost);
  
  if (cuda_status != hipSuccess)
    {
      printf("Failed to copy data from device!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}

/* Function to call the matrix vector multiply kernel */
extern "C" int matrix_vector_multiplies(int *n_mat,
					int *n_row,
					int *n_col,
					void **d_lhs_vector,
					void **d_matrix,
					void **d_rhs_vector)
{
  /* Syntax <<<NumBlocks, ThreadsPerBlock>>> */
  /* Max no threads per dimension per block 1024 */
  /* Max size of grid in each dimension 65535 */
  /* Warp size 32 */
  int NumBlocks;
  int ThreadsPerBlock;
  hipError_t cuda_status;

  int version = 2;

  if (version == 1)
    {
      ThreadsPerBlock = 1024;
      NumBlocks = (*n_mat * *n_row)/ThreadsPerBlock;
      
      if ( (*n_mat * *n_row)%ThreadsPerBlock != 0)
	NumBlocks += 1;
      
      /* Launch kernel */
      MultiMatVecMultiply1<<<NumBlocks, ThreadsPerBlock>>>
	(*n_mat,
	 *n_row,
	 *n_col,
	 (double *)(*d_lhs_vector),
	 (double *)(*d_matrix),
	 (double *)(*d_rhs_vector));
    }
  else
    {
      ThreadsPerBlock = 60;
      NumBlocks = (*n_mat * *n_row)/ThreadsPerBlock;
      
      if ( (*n_mat * *n_row)%ThreadsPerBlock != 0)
	NumBlocks += 1;
      
      /* Launch kernel */
      MultiMatVecMultiply2<<<NumBlocks, ThreadsPerBlock>>>
	(*n_mat,
	 *n_row,
	 *n_col,
	 (double *)(*d_lhs_vector),
	 (double *)(*d_matrix),
	 (double *)(*d_rhs_vector));
      
    }


  /* Test for error at kernel launch */
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess)
    {
      printf("Kernel launch failure!\n");
      return EXIT_FAILURE;
    }
  
  /* Ensure synchronisation */
  /* Note: this should pick up kernel errors */
  cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess)
    {
      printf("Streams failed to synchronise!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}
 
 
