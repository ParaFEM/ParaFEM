
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hipblas.h>

/* Kernel definitions */

extern __shared__ float work_array[];

/* 
  This kernel assumes the matrix is stored column wise

  The kernel uses 2D thread blocks to try to improve performance
*/
__global__ void MultiMatVecMultiply3(int n_mat,
				     int n_row,
				     int n_col,
				     double* lhs_vector,
				     double* matrix,
				     double* rhs_vector)
{
  int matrix_id;
  int global_row_id;
  int row_id;
  int j;
  double tmp;

  /* Get the global index that corresponds to a row of some matrix */
  global_row_id =  threadIdx.x + blockIdx.x * blockDim.x;

  if (global_row_id < n_mat*n_row)
    {
      /* get matrix id */
      matrix_id = global_row_id/n_row;
      
      /* Get local row id */
      row_id = global_row_id%n_row;
      
      /* Variable to store partial row value */
      tmp = 0.0;
      
      /* Each thread loops through a part of the row */
      for (j=threadIdx.y; j<n_col; j+= blockDim.y)
	{
	  tmp +=
	    lhs_vector[j+matrix_id*n_col] *
	    matrix[row_id+j*n_row+matrix_id*n_col*n_row];
	}

      /* Put values in shared memory array */
      work_array[threadIdx.x + threadIdx.y*blockDim.x] = tmp;

      /* Sync threads */
      __syncthreads();

      /* Reduce array values */
      j = blockDim.y;

      while ( j > 1 )
	{
	  j >>= 1; /* Divide j by 2 */

	  if ( threadIdx.y < j )
	    {
	      work_array[threadIdx.x + threadIdx.y*blockDim.x]
		+= work_array[threadIdx.x + (threadIdx.y+j)*blockDim.x];

	      /* Sync threads */
	      __syncthreads();
	    }
	}
      
      /* Copy reduced value to result vector */
      rhs_vector[global_row_id] = work_array[threadIdx.x];
    }
}


/* This kernel used shared memory to store the lhs vector
   and assumes:

   - the matrix is stored column wise 
   - the 1D thread block is of size 60
   - 60x60 matrix

*/
__global__ void MultiMatVecMultiply2(int n_mat,
				     int n_row,
				     int n_col,
				     double* lhs_vector,
				     double* matrix,
				     double* rhs_vector)
{
  int local_thread_id;
  int matrix_id;
  int global_thread_id;
  int row_id;
  int j; 
  double tmp;
  
  /* Get 1D thread index */
  local_thread_id = threadIdx.x;
  
  /* Global thread index */
  global_thread_id = local_thread_id + blockIdx.x * blockDim.x; 

  /* get matrix id */
  matrix_id = global_thread_id/n_row;

  /* Shared memory to store copy of lhs vector */
  __shared__ double lhs_vector_shared[60];
  
  /* Get row id */
  row_id = global_thread_id%n_row; 
  
  /* Load values into shared vector */
  if (local_thread_id < 60)
    lhs_vector_shared[local_thread_id] = 
      lhs_vector[matrix_id*n_col+local_thread_id];

  /* Sunc threads */
  __syncthreads();

  if (global_thread_id < n_mat*n_row)
    {
      tmp = 0.0;
      for (j=0; j<n_col; j++)
	{
	  tmp +=
	    lhs_vector_shared[j] * 
	    matrix[j*n_row+row_id+matrix_id*n_col*n_row];
	}
      rhs_vector[global_thread_id] = tmp;
    }
}

/* 
  This kernel assumes matrix is stored column wise

  This is a naive implementation using 1D thread blocks, 
  each thread multiples one matrix row by 1 vector
*/
__global__ void MultiMatVecMultiply1(int n_mat,
				     int n_row,
				     int n_col,
				     double* lhs_vector,
				     double* matrix,
				     double* rhs_vector)
{
  int matrix_id;
  int global_thread_id;
  int row_id;
  int j; 
  double tmp;
  
  global_thread_id =  threadIdx.x + blockIdx.x * blockDim.x;  
  
  if (global_thread_id < n_mat*n_row)
    {
      
      /* get matrix id */
      matrix_id = global_thread_id/n_row;
      
      /* Get row id */
      row_id = global_thread_id%n_row; 
      
      tmp = 0.0;
      for (j=0; j<n_col; j++)
	{
	  tmp +=
	    lhs_vector[j+matrix_id*n_col] * 
	    matrix[row_id+j*n_row+matrix_id*n_col*n_row];
	}
      rhs_vector[global_thread_id] = tmp;
    }
}


/* Helper functions */

/* Function to call cudaSetDevice */
extern "C" int set_gpu(const int *device_id)
{
  hipError_t cuda_status;

  cuda_status = hipSetDevice(*device_id);

  if (cuda_status != hipSuccess)
    {
      printf("Failed to set device!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}

/* Function to allocate memory on device */
extern "C" int allocate_memory_on_gpu(const int *n_elements, 
				      const int *element_size,
				      void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipMalloc(device_pointer, *element_size * *n_elements);
  
  if (cuda_status != hipSuccess)
    {
      printf("Device memory failed to allocate!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}

/* Function to free memory on device */
extern "C" int free_memory_on_gpu(void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipFree(*device_pointer);
  
  if (cuda_status != hipSuccess)
    {
      printf("Device memory failed to deallocate!\n");
      return EXIT_FAILURE;
    }
  
  return EXIT_SUCCESS;
}

/* Function to copy data to the gpu */
extern "C" int copy_data_to_gpu(const int *n_elements,
				const int *element_size,
				const void *host_data,
				void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipMemcpy(*device_pointer,
			   host_data,
			   *n_elements * *element_size,
			   hipMemcpyHostToDevice);
  
  if (cuda_status != hipSuccess)
    {
      printf("Failed to copy data to device!\n");
      return EXIT_FAILURE;
    }
  
  return EXIT_SUCCESS;
 }

/* Function to copy data from the gpu */
extern "C" int copy_data_from_gpu(const int *n_elements, 
				  const int *element_size,
				  void *host_data,
				  void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipMemcpy(host_data,
			   *device_pointer,
			   *n_elements * *element_size,
			   hipMemcpyDeviceToHost);
  
  if (cuda_status != hipSuccess)
    {
      printf("Failed to copy data from device!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}

/* Function to call the matrix vector multiply kernel */
extern "C" int matrix_vector_multiplies(int *n_mat,
					int *n_row,
					int *n_col,
					void **d_lhs_vector,
					void **d_matrix,
					void **d_rhs_vector)
{
  /* Max no threads per dimension per block 1024 */
  /* Max size of grid in each dimension 65535 */
  /* Warp size 32 */
  int NumBlocks;
  int ThreadsPerBlock_1D;
  dim3 ThreadsPerBlock_2D;
  hipError_t cuda_status;

  int method = 1;

  size_t shared_mem_size;

  if (method == 1)
    {
      ThreadsPerBlock_1D = 128;
      NumBlocks = (*n_mat * *n_row)/ThreadsPerBlock_1D;
      
      if ( (*n_mat * *n_row)%ThreadsPerBlock_1D != 0)
	NumBlocks += 1;
      
      /* Launch kernel */
      MultiMatVecMultiply1<<<NumBlocks, ThreadsPerBlock_1D>>>
	(*n_mat,
	 *n_row,
	 *n_col,
	 (double *)(*d_lhs_vector),
	 (double *)(*d_matrix),
	 (double *)(*d_rhs_vector));
    }
  else if (method == 2)
    {
      ThreadsPerBlock_1D = *n_row;
      NumBlocks = *n_mat;
      
      /* Launch kernel */
      MultiMatVecMultiply2<<<NumBlocks, ThreadsPerBlock_1D>>>
	(*n_mat,
	 *n_row,
	 *n_col,
	 (double *)(*d_lhs_vector),
	 (double *)(*d_matrix),
	 (double *)(*d_rhs_vector));
      
    }
  else
    {
      NumBlocks = *n_mat;
      
      ThreadsPerBlock_2D.x = *n_row;
      ThreadsPerBlock_2D.y = 2;
      ThreadsPerBlock_2D.z = 1;
      
      shared_mem_size = sizeof(double) * 
	ThreadsPerBlock_2D.x * ThreadsPerBlock_2D.y;

      MultiMatVecMultiply3
	<<<NumBlocks, ThreadsPerBlock_2D, shared_mem_size>>>
	(*n_mat,
	 *n_row,
	 *n_col,
	 (double *)(*d_lhs_vector),
	 (double *)(*d_matrix),
	 (double *)(*d_rhs_vector));
    }


  /* Test for error at kernel launch */
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess)
    {
      printf("Kernel launch failure!\n");
      return EXIT_FAILURE;
    }
  
  /* Ensure synchronisation */
  /* Note: this should pick up kernel errors */
  cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess)
    {
      printf("Streams failed to synchronise!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}
 
 
