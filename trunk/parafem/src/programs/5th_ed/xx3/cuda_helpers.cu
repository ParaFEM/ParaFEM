
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hipblas.h>

/* Kernel definitions */

extern __shared__ float work_array[];

/* 
  This kernel assumes matrix is stored column wise

  This attempts 2D thread blocks
*/
__global__ void MultiMatVecMultiply3(int n_mat,
				     int n_row,
				     int n_col,
				     double* lhs_vector,
				     double* matrix,
				     double* rhs_vector)
{
  int global_row_id;
  int matrix_id;
  int row_id;
  int j;
  double tmp;

  /* Get the global index that corresponds to a row of some matrix */
  global_row_id =  threadIdx.x + blockIdx.x * blockDim.x;

  if (global_row_id < n_mat*n_row)
    {
      /* get matrix id */
      matrix_id = global_row_id/n_row;
      
      /* Get local row id */
      row_id = global_row_id%n_row;
      
      /* Variable to store partial row value */
      tmp = 0.0;
      
      /* Each thread loop through a part of the row */
      for (j=threadIdx.y; j<n_col; j+= blockDim.y)
	{
	  tmp +=
	    lhs_vector[j+matrix_id*n_col] *
	    matrix[row_id+j*n_row+matrix_id*n_col*n_row];
	}

      /* Put values in shared memory array */
      work_array[threadIdx.x + threadIdx.y*n_row] = tmp;

      /* Sync threads */
      __syncthreads();

      /* Reduce array values */
      j = threadIdx.y;

      while ( j>1 )
	{
	  j >>= 1; /* Divide j by 2 */

	  if (threadIdx.y<j)
	    {
	      work_array[threadIdx.x + threadIdx.y*n_row]
		+= work_array[threadIdx.x + threadIdx.y*(n_row+j)];

	      /* Sync threads */
	      __syncthreads();
	    }
	}
      
      /* Copy reduced value to result vector */
      rhs_vector[global_row_id] = work_array[j];
    }
}


/* This kernel used shared memory but assumes:

   - the matrix is stored column wise 
   - the 1D thread block is of size 60
   - 60x60 matrix

*/
__global__ void MultiMatVecMultiply2(int n_mat,
				     int n_row,
				     int n_col,
				     double* lhs_vector,
				     double* matrix,
				     double* rhs_vector)
{
  int local_thread_id;
  int block_id;
  int matrix_id;
  int global_thread_id;
  int row_id;
  int j; 
  double tmp;
  
  /* Get 1D thread index */
  local_thread_id = threadIdx.x;
  
  /* Get 1D block index */
  block_id = blockIdx.x;

  /* Global thread index */
  global_thread_id = local_thread_id + block_id * blockDim.x; 

  /* get matrix id */
  matrix_id = global_thread_id/n_row;

  /* Shared memory to store copy of lhs vector */
  __shared__ double lhs_vector_shared[60];
 /*  double* lhs_vector_shared = (double*)work_array; */
  
  /* Get row id */
  row_id = global_thread_id%n_row; 
  
  /* Load values into shared vector */
  if (local_thread_id < 60)
    lhs_vector_shared[local_thread_id] = 
      lhs_vector[matrix_id*n_col+local_thread_id];

  /* Sunc threads */
  __syncthreads();

  if (global_thread_id < n_mat*n_row)
    {
      tmp = 0.0;
      for (j=0; j<n_col; j++)
	{
	  tmp +=
	    lhs_vector_shared[j] * 
	    matrix[j*n_row+row_id+matrix_id*n_col*n_row];
	}
      rhs_vector[global_thread_id] = tmp;
    }
}

/* 
  This kernel assumes matrix is stored column wise

  This is a naive implementation, no attempt at optimisation
*/
__global__ void MultiMatVecMultiply1(int n_mat,
				     int n_row,
				     int n_col,
				     double* lhs_vector,
				     double* matrix,
				     double* rhs_vector)
{
  int local_thread_id;
  int block_id;
  int matrix_id;
  int global_thread_id;
  int row_id;
  int j; 
  double tmp;
  
  /* Get 1D thread index */
  local_thread_id = threadIdx.x;
  
  /* Get 1D block index */
  block_id = blockIdx.x;

  global_thread_id = local_thread_id + block_id * blockDim.x;  
  
  if (global_thread_id < n_mat*n_row)
    {
      
      /* get matrix id */
      matrix_id = global_thread_id/n_row;
      
      /* Get row id */
      row_id = global_thread_id%n_row; 
      
      tmp = 0.0;
      for (j=0; j<n_col; j++)
	{
	  tmp +=
	    lhs_vector[j+matrix_id*n_col] * 
	    matrix[row_id+j*n_row+matrix_id*n_col*n_row];
	}
      rhs_vector[global_thread_id] = tmp;
    }
}


/* Helper functions */

/* Function to allocate memory on device */
extern "C" int allocate_memory_on_gpu(const int *n_elements, 
				      const int *element_size,
				      void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipMalloc(device_pointer, *element_size * *n_elements);
  
  if (cuda_status != hipSuccess)
    {
      printf("Device memory failed to allocate!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}

/* Function to free memory on device */
extern "C" int free_memory_on_gpu(void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipFree(*device_pointer);
  
  if (cuda_status != hipSuccess)
    {
      printf("Device memory failed to deallocate!\n");
      return EXIT_FAILURE;
    }
  
  return EXIT_SUCCESS;
}

/* Function to copy data to the gpu */
extern "C" int copy_data_to_gpu(const int *n_elements,
				const int *element_size,
				const void *host_data,
				void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipMemcpy(*device_pointer,
			   host_data,
			   *n_elements * *element_size,
			   hipMemcpyHostToDevice);
  
  if (cuda_status != hipSuccess)
    {
      printf("Failed to copy data to device!\n");
      return EXIT_FAILURE;
    }
  
  return EXIT_SUCCESS;
 }

/* Function to copy data from the gpu */
extern "C" int copy_data_from_gpu(const int *n_elements, 
				  const int *element_size,
				  void *host_data,
				  void **device_pointer)
{
  hipError_t cuda_status;
  
  cuda_status = hipMemcpy(host_data,
			   *device_pointer,
			   *n_elements * *element_size,
			   hipMemcpyDeviceToHost);
  
  if (cuda_status != hipSuccess)
    {
      printf("Failed to copy data from device!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}

/* Function to call the matrix vector multiply kernel */
extern "C" int matrix_vector_multiplies(int *n_mat,
					int *n_row,
					int *n_col,
					void **d_lhs_vector,
					void **d_matrix,
					void **d_rhs_vector)
{
  /* Syntax <<<NumBlocks, ThreadsPerBlock>>> */
  /* Max no threads per dimension per block 1024 */
  /* Max size of grid in each dimension 65535 */
  /* Warp size 32 */
  int NumBlocks;
  int ThreadsPerBlock_1D;
  dim3 ThreadsPerBlock_2D;
  hipError_t cuda_status;

  int method = 2;

  size_t shared_mem_size;

  if (method == 3)
    {
      ThreadsPerBlock_1D = 1024;
      NumBlocks = (*n_mat * *n_row)/ThreadsPerBlock_1D;
      
      if ( (*n_mat * *n_row)%ThreadsPerBlock_1D != 0)
	NumBlocks += 1;
      
      /* Launch kernel */
      MultiMatVecMultiply1<<<NumBlocks, ThreadsPerBlock_1D>>>
	(*n_mat,
	 *n_row,
	 *n_col,
	 (double *)(*d_lhs_vector),
	 (double *)(*d_matrix),
	 (double *)(*d_rhs_vector));
    }
  else if (method == 2)
    {
      ThreadsPerBlock_1D = *n_row;
      NumBlocks = *n_mat; /* (*n_mat * *n_row)/ThreadsPerBlock_1D;
      
      if ( (*n_mat * *n_row)%ThreadsPerBlock_1D != 0)
      NumBlocks += 1; */
      
      /* Launch kernel */
      MultiMatVecMultiply2<<<NumBlocks, ThreadsPerBlock_1D>>>
	(*n_mat,
	 *n_row,
	 *n_col,
	 (double *)(*d_lhs_vector),
	 (double *)(*d_matrix),
	 (double *)(*d_rhs_vector));
      
    }
  else
    {
      NumBlocks = *n_mat;
      
      ThreadsPerBlock_2D.x = *n_row;
      ThreadsPerBlock_2D.y = 2;
      ThreadsPerBlock_2D.y = 1;
      
      shared_mem_size = sizeof(*d_matrix) * 
	ThreadsPerBlock_2D.x * ThreadsPerBlock_2D.y;

      MultiMatVecMultiply3
	<<<NumBlocks, ThreadsPerBlock_2D, shared_mem_size>>>
	(*n_mat,
	 *n_row,
	 *n_col,
	 (double *)(*d_lhs_vector),
	 (double *)(*d_matrix),
	 (double *)(*d_rhs_vector));
    }


  /* Test for error at kernel launch */
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess)
    {
      printf("Kernel launch failure!\n");
      return EXIT_FAILURE;
    }
  
  /* Ensure synchronisation */
  /* Note: this should pick up kernel errors */
  cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess)
    {
      printf("Streams failed to synchronise!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}
 
 
