
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hipblas.h>


/* Helper function */

/* Function to call cudaSetDevice */
extern "C" int set_gpu(const int *device_id)
{
  hipError_t cuda_status;

  cuda_status = hipSetDevice(*device_id);

  if (cuda_status != hipSuccess)
    {
      printf("Failed to set device!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}

