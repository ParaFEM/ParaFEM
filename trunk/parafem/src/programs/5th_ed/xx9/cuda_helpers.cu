
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hipblas.h>


/* Helper function */

/* Function to call cudaSetDevice */
extern "C" int set_gpu(const int *device_id)
{
  hipError_t cuda_status;

  cuda_status = hipSetDevice(*device_id);

  if (cuda_status != hipSuccess)
    {
      printf("Failed to set device!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}

/* Function to call cudaDeviceSynchronize */
extern "C" int sync_gpu()
{
  hipError_t cuda_status;  

  cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess)
    {
      printf("Streams failed to synchronise!\n");
      return EXIT_FAILURE;
    }

  return EXIT_SUCCESS;
}
